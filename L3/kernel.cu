#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""

#include <fstream>
#include <iostream>
#include <vector>
#include <stdio.h>

#include "json/single_include/nlohmann/json.hpp"

using json = nlohmann::json;

struct Car {
    char name[50];
    int fuel_tank_size;
    float fuel_efficiency;
};

struct Result {
    char data[60];
};

std::vector<Car> readFile(const std::string& filename) {
    std::ifstream file(filename);
    if (!file.is_open()) {
		std::cout << "Could not open file: " << filename << std::endl;
        exit(EXIT_FAILURE);
	}

    nlohmann::json jsonData;
    file >> jsonData;
    file.close();

    std::vector<Car> cars;
    for (const auto& carData : jsonData["cars"]) {
        Car car;
        strcpy(car.name, carData["name"].get<std::string>().c_str());
        car.fuel_tank_size = carData["fuel_tank_size"].get<int>();
        car.fuel_efficiency = carData["fuel_efficiency"].get<float>();
        cars.push_back(car);
    }

    return cars;
}

void printResults(const Result* results, int size) {
    int counter = 0;
    for (int i = 0; i < size; i++) {
        if (results[i].data[0] != '\0') {
            std::cout << results[i].data << std::endl;
            counter++;
        }
    }
    std::cout << "Number of results: " << counter << std::endl;
}

void writeResultsToFile(const Result* results, int size, std::string filename) {
	std::ofstream file(filename);
	if (!file.is_open()) {
		std::cout << "Could not open file: " << filename << std::endl;
		exit(EXIT_FAILURE);
	}

	int counter = 0;
	for (int i = 0; i < size; i++) {
		if (results[i].data[0] != '\0') {
			file << results[i].data << std::endl;
			counter++;
		}
	}
    file << "Number of results: " << counter;
	file.close();

    std::cout << "Successfully wrote results to file: " << filename << std::endl;
}

void writeResultsToConsole(const Result* results, int size) {
    char ans;
    
    std::cout << "Do you want to see the results in the console? (Enter 'Y' for Yes, 'N' for No): ";
    std::cin >> ans;
    ans = std::toupper(ans);

    if (ans != 'Y') {
        return;
    }

    int counter = 0;
    for (int i = 0; i < size; i++) {
        if (results[i].data[0] != '\0') {
            std::cout << results[i].data << std::endl;
            counter++;
        }
    }

    std::cout << "Number of results: " << counter << std::endl;
}

void checkCudaDevice() {
    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    if (deviceCount == 0) {
		std::cout << "There is no CUDA device" << std::endl;
		exit(EXIT_FAILURE);
	}

    for (int i = 0; i < deviceCount; ++i) {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, i);
        std::cout << "Device " << i << ": " << deviceProp.name << std::endl;
    }
}

__device__ char convertToRating(float efficiency) {
    if (efficiency > 30.0f) {
        return 'A';
    }
    else if (efficiency >= 25.0f) {
        return 'B';
    } 
    else if (efficiency >= 20.0f) {
		return 'C';
	}
	else if (efficiency >= 15.0f) {
		return 'D';
	}
	else {
		return 'E';
	}
}

__global__ void filterAndSortCars(Car* cars, int size, int tankSizeThreshold, Result* results, int resultsSize) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < size) {
        if (cars[index].fuel_tank_size > tankSizeThreshold) {

            Result result;

            // Copy name to result.data
            const char* namePtr = cars[index].name;
            char* resultPtr = result.data;
            while (*namePtr != '\0') {
				*resultPtr = *namePtr;
				++namePtr;
				++resultPtr;
			}

            *resultPtr = '-';
            ++resultPtr;

            // Convert fuel_efficiency to rating and append to result.data
            *resultPtr = convertToRating(cars[index].fuel_efficiency);
            ++resultPtr;

            // Append fuel tank size to result.data
            int tankSize = cars[index].fuel_tank_size;
            if (tankSize >= 100) {
                *resultPtr = '1';
                ++resultPtr;
                *resultPtr = '0' + (tankSize % 100) / 10;
                ++resultPtr;
                *resultPtr = '0' + tankSize % 10;  // Increment resultPtr for the last digit
                ++resultPtr;
            }
            else {
                *resultPtr = '0' + tankSize / 10;
                ++resultPtr;
                *resultPtr = '0' + tankSize % 10;  // Increment resultPtr for the last digit
                ++resultPtr;
            }

            // Find the first free slot in the results array
            int resultIndex = index % resultsSize;

            while (atomicCAS((int*)&results[resultIndex].data[0], 0, 1) != 0) {
                resultIndex = (resultIndex + 1) % resultsSize;
            }

            results[resultIndex] = result;
        }
    }
}

int main() {
    // Check and print CUDA device information
    checkCudaDevice();

    std::string outputFile = "IFF-1-1_PaulaviciusK_L3_res.txt";

    // User input to select file to read
    std::string inputFile;
    std::cout << "Select file (1 to 3): ";
    std::cin >> inputFile;
    if (inputFile == "1") {
        inputFile = "IFF-1-1_PaulaviciusK_L3_dat_1.json";
    }
    else if (inputFile == "2") {
        inputFile = "IFF-1-1_PaulaviciusK_L3_dat_2.json";
    }
    else if (inputFile == "3") {
        inputFile = "IFF-1-1_PaulaviciusK_L3_dat_3.json";
    }
    else {
        std::cout << "Invalid input" << std::endl;
        exit(EXIT_FAILURE);
    }

    std::cout << "Reading file: " << inputFile << std::endl;

    // Host variables
    std::vector<Car> hostCars = readFile(inputFile);
    const int dataSize = hostCars.size();
    const int fuelTankSizeThreshold = 60;

    // Device variables
    Car* deviceCars;
    Result* hostResults = new Result[dataSize];
    Result* deviceResults;

    // Ensure at least two blocks and the number of threads per block is a multiple of 32
    const int threadsPerBlock = 32 * 2;
    const int blocksPerGrid = (dataSize + threadsPerBlock - 1) / threadsPerBlock;

    // Allocate memory on GPU
    hipMalloc((void**)&deviceCars, dataSize * sizeof(Car));
    hipMalloc((void**)&deviceResults, dataSize * sizeof(Result));

    // Copy data from host to device
    hipMemcpy(deviceCars, hostCars.data(), dataSize * sizeof(Car), hipMemcpyHostToDevice);

    // Initialize results array on host
    memset(hostResults, 0, dataSize * sizeof(Result));

    // Copy results array from host to device
    hipMemcpy(deviceResults, hostResults, dataSize * sizeof(Result), hipMemcpyHostToDevice);

    // Launch CUDA kernel
    filterAndSortCars<<<blocksPerGrid, threadsPerBlock>>>(deviceCars, dataSize, fuelTankSizeThreshold, deviceResults, dataSize);
    hipDeviceSynchronize(); // Add synchronization to ensure the kernel is finished before copying data back

    // Copy data from device to host
    hipMemcpy(hostResults, deviceResults, dataSize * sizeof(Result), hipMemcpyDeviceToHost);

    // Print results
    writeResultsToFile(hostResults, dataSize, outputFile);

    // Ask to write to console
    writeResultsToConsole(hostResults, dataSize);

    // Free memory on GPU and host
    hipFree(deviceCars);
    hipFree(deviceResults);

    return 0;
}
